
#include <hip/hip_runtime.h>
#include <iostream>
#include <hip/hip_runtime_api.h>
#include <chrono>

//#define DEBUG_DEV

#ifdef DEBUG_DEV
#define getErrorCuda(command)\
        command;\
        cudaDeviceSynchronize();\
        cudaThreadSynchronize();\
        if (cudaPeekAtLastError() != cudaSuccess){\
            std::cout << #command << " : " << cudaGetErrorString(cudaGetLastError())\
             << " in file " << __FILE__ << " at line " << __LINE__ << std::endl;\
            exit(1);\
        }
#endif
#ifndef DEBUG_DEV
#define getErrorCuda(command) command;
#endif

__constant__ float const_stencilWeight[21];


// base case
__global__ void stencil(float *src, float *dst, int size, float *stencilWeight)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    idx += 11;
    if (idx >= size)
        return;
    float out = 0;
    #pragma unroll
    for(int i = -10;i <= 10; i++)
    {
        out += src[idx+i] * stencilWeight[i+10];
    }
    dst[idx] = out;
}

// read only cache stencil coefficients
__global__ void stencilReadOnly1(float *src, float *dst, int size, float* stencilWeight)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    idx += 11;
    if (idx >= size)
        return;
    float out = 0;
    #pragma unroll
    for(int i = -10;i <= 10; i++)
    {
        out += src[idx+i] * __ldg(&stencilWeight[i+10]);
    }
    dst[idx] = out;
}

// read only data
__global__ void stencilReadOnly2(float *src, float *dst, int size, float* stencilWeight)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    idx += 11;
    if (idx >= size)
        return;
    float out = 0;
    #pragma unroll
    for(int i = -10;i <= 10; i++)
    {
        out += __ldg(&src[idx+i]) * stencilWeight[i+10];
    }
    dst[idx] = out;
}

// read only coefficients and data
__global__ void stencilReadOnly3(float *src, float *dst, int size, float* stencilWeight)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    idx += 11;
    if (idx >= size)
        return;
    float out = 0;
    #pragma unroll
    for(int i = -10;i <= 10; i++)
    {
        out += __ldg(&src[idx+i]) * __ldg(&stencilWeight[i+10]);
    }
    dst[idx] = out;
}

// constat memory coefficients
__global__ void stencilConst1(float *src, float *dst, int size)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    idx += 11;
    if (idx >= size)
        return;
    float out = 0;
    #pragma unroll
    for(int i = -10;i <= 10; i++)
    {
        out += src[idx+i] * const_stencilWeight[i+10];
    }
    dst[idx] = out;
}

// constant memory coefficients and data through read only cache
__global__ void stencilConst2(float *src, float *dst, int size)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    idx += 11;
    if (idx >= size)
        return;
    float out = 0;
    #pragma unroll
    for(int i = -10;i <= 10; i++)
    {
        out += __ldg(&src[idx+i]) * const_stencilWeight[i+10];
    }
    dst[idx] = out;
}

// constant memory coefficients and data from shared
__global__ void stencilShared1(float *src, float *dst, int size)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    __shared__ float buffer[1024+21];
    for(int i = threadIdx.x; i < 1024+21; i = i + 1024)
    {
        buffer[i] = src[idx+i];
    }
    idx += 11;
    if (idx >= size)
        return;
   
    __syncthreads();
    float out = 0;
    #pragma unroll
    for(int i = -10;i <= 10; i++)
    {
        out += buffer[threadIdx.x+10+i] * const_stencilWeight[i+10];
    }
    dst[idx] = out;
}

// constant memory coefficients and data from shared thorugh read only
__global__ void stencilShared2(float *src, float *dst, int size)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    __shared__ float buffer[1024+21];
    for(int i = threadIdx.x; i < 1024+21; i = i + 1024)
    {
        buffer[i] = __ldg(&src[idx+i]);
    }
    idx += 11;
    if (idx >= size)
        return;
   
    __syncthreads();
    float out = 0;
    #pragma unroll
    for(int i = -10;i <= 10; i++)
    {
        out += buffer[threadIdx.x+10+i] * const_stencilWeight[i+10];
    }
    dst[idx] = out;
}

void verify(float *arr, float *corr, int count)
{
    for(int i = 40; i < count; i++)
    {
        if(arr[i] != corr[i])
        {
            std::cout << "error verifying resutls" << std::endl;
            exit(1);
        }
    }
}

int main()
{
    float *a;
    float *b;
    float *bOut;
    float *bCorr;
    float *weights;
    getErrorCuda(hipMalloc(&a, sizeof(float)*102400000));
    getErrorCuda(hipMalloc(&b, sizeof(float)*102400000));
    getErrorCuda(hipHostMalloc(&bOut, sizeof(float)*102400000, hipHostMallocDefault));
    getErrorCuda(hipMallocManaged(&bCorr, sizeof(float)*102400000));
    getErrorCuda(hipMallocManaged(&weights, sizeof(float)*21));

    hipDeviceSynchronize();   

    for(int i = 0; i < 102400000;i++)
    {
        //a[i] = 0;
        //b[i] = 0;
        bCorr[i] = 0;
    }
    hipDeviceSynchronize();   
   
    int blockSize = 1024;
    int blocks = 10000;
    for(int i = 0; i < 21;i++)
        weights[i] = i-10;
   
   
    hipDeviceSynchronize();   
   
    hipMemcpyToSymbol(HIP_SYMBOL(const_stencilWeight), weights, sizeof(float)*21);

    stencil<<<blocks, blockSize>>>(a, bCorr, 10240000-11, weights);
    hipDeviceSynchronize();   

    stencil<<<blocks, blockSize>>>(a, b, 10240000-11, weights);
    hipDeviceSynchronize();
    getErrorCuda(hipMemcpy(bOut, b, sizeof(float)*10240000, hipMemcpyDefault));
    verify(bOut, bCorr, 1000);

    hipSetDevice(0);


    float minTime = 10000;
    for(int i  = 0; i < 100; i++)
    {
        std::chrono::time_point<std::chrono::system_clock> start, end;
        start = std::chrono::system_clock::now();
        stencil<<<blocks, blockSize>>>(a, b, 10240000-11, weights);
        hipDeviceSynchronize();   
        end = std::chrono::system_clock::now();
       
        getErrorCuda(hipMemcpy(bOut, b, sizeof(float)*10240000, hipMemcpyDefault));
        verify(bOut, bCorr, 1000); 

        std::chrono::duration<float> elapsed_seconds = end-start;
        minTime = std::min(elapsed_seconds.count(), minTime);
    }
    std::cout << "Non optimized time " << (blockSize*blocks)/minTime << " elem/s" << " Read BW " << (21*blockSize*blocks*sizeof(float)/1000.0/1000.0/1000.0 )/minTime << " GB/s" <<   std::endl;
    minTime = 10000;
    std::cout << std::endl;

    for(int i  = 0; i < 100; i++)
    {
        hipDeviceSynchronize(); 
        std::chrono::time_point<std::chrono::system_clock> start, end;
        start = std::chrono::system_clock::now();
        stencilReadOnly1<<<blocks, blockSize>>>(a, b, 10240000-11, weights);
        hipDeviceSynchronize(); 
        end = std::chrono::system_clock::now();
       
        getErrorCuda(hipMemcpy(bOut, b, sizeof(float)*10240000, hipMemcpyDefault));
        verify(bOut, bCorr, 1000); 
       
        std::chrono::duration<float> elapsed_seconds = end-start;
        minTime = std::min(elapsed_seconds.count(), minTime);
    }
    std::cout << "read only cache stencil coefficients time " <<(blockSize*blocks)/minTime << " elem/s" << " Read BW " << (21*blockSize*blocks*sizeof(float)/1000.0/1000.0/1000.0 )/minTime << " GB/s" <<   std::endl;
    minTime = 10000;
    for(int i  = 0; i < 100; i++)
    {
        hipDeviceSynchronize(); 
        std::chrono::time_point<std::chrono::system_clock> start, end;
        start = std::chrono::system_clock::now();
        stencilReadOnly2<<<blocks, blockSize>>>(a, b, 10240000-11, weights);
        hipDeviceSynchronize(); 
        end = std::chrono::system_clock::now();
       
        getErrorCuda(hipMemcpy(bOut, b, sizeof(float)*10240000, hipMemcpyDefault));
        verify(bOut, bCorr, 1000); 
       
        std::chrono::duration<float> elapsed_seconds = end-start;
        minTime = std::min(elapsed_seconds.count(), minTime);
    }
    std::cout << "read only data time " << (blockSize*blocks)/minTime << " elem/s" << " Read BW " << (21*blockSize*blocks*sizeof(float)/1000.0/1000.0/1000.0 )/minTime << " GB/s" <<   std::endl;
    minTime = 10000;
        for(int i  = 0; i < 100; i++)
    {
        hipDeviceSynchronize(); 
        std::chrono::time_point<std::chrono::system_clock> start, end;
        start = std::chrono::system_clock::now();
        stencilReadOnly3<<<blocks, blockSize>>>(a, b, 10240000-11, weights);
        hipDeviceSynchronize(); 
        end = std::chrono::system_clock::now();
       
        getErrorCuda(hipMemcpy(bOut, b, sizeof(float)*10240000, hipMemcpyDefault));
        verify(bOut, bCorr, 1000); 
       
        std::chrono::duration<float> elapsed_seconds = end-start;
        minTime = std::min(elapsed_seconds.count(), minTime);
    }
    std::cout << "read only coefficients and data time " << (blockSize*blocks)/minTime << " elem/s" << " Read BW " << (21*blockSize*blocks*sizeof(float)/1000.0/1000.0/1000.0 )/minTime << " GB/s" <<   std::endl;
    minTime = 10000;

    std::cout << std::endl;

        for(int i  = 0; i < 100; i++)
    {
        hipDeviceSynchronize(); 
       
        std::chrono::time_point<std::chrono::system_clock> start, end;
        start = std::chrono::system_clock::now();
        stencilConst1<<<blocks, blockSize>>>(a, b, 10240000);
        hipDeviceSynchronize();   
        end = std::chrono::system_clock::now();

        getErrorCuda(hipMemcpy(bOut, b, sizeof(float)*10240000, hipMemcpyDefault));
        verify(bOut, bCorr, 1000); 
        std::chrono::duration<float> elapsed_seconds = end-start;
        minTime = std::min(elapsed_seconds.count(), minTime);
    }
    std::cout << "constant memory coefficients " << (blockSize*blocks)/minTime << " elem/s" << " Read BW " << (21*blockSize*blocks*sizeof(float)/1000.0/1000.0/1000.0 )/minTime << " GB/s" <<   std::endl;

    minTime = 10000;


        for(int i  = 0; i < 100; i++)
    {
        hipDeviceSynchronize(); 
       
        std::chrono::time_point<std::chrono::system_clock> start, end;
        start = std::chrono::system_clock::now();
        stencilConst2<<<blocks, blockSize>>>(a, b, 10240000);
        hipDeviceSynchronize();   
        end = std::chrono::system_clock::now();

        getErrorCuda(hipMemcpy(bOut, b, sizeof(float)*10240000, hipMemcpyDefault));
        verify(bOut, bCorr, 1000); 
        std::chrono::duration<float> elapsed_seconds = end-start;
        minTime = std::min(elapsed_seconds.count(), minTime);
    }
    std::cout << "constant memory coefficients and data through read only cache time " << (blockSize*blocks)/minTime << " elem/s" << " Read BW " << (21*blockSize*blocks*sizeof(float)/1000.0/1000.0/1000.0 )/minTime << " GB/s" <<   std::endl;
    std::cout << std::endl;


    minTime = 10000;
            for(int i  = 0; i < 100; i++)
    {
        hipDeviceSynchronize(); 
       
        std::chrono::time_point<std::chrono::system_clock> start, end;
        start = std::chrono::system_clock::now();
        stencilShared1<<<blocks, blockSize>>>(a, b, 10240000);
        hipDeviceSynchronize();   
        end = std::chrono::system_clock::now();
       
        getErrorCuda(hipMemcpy(bOut, b, sizeof(float)*10240000, hipMemcpyDefault));
        verify(bOut, bCorr, 1000); 
        std::chrono::duration<float> elapsed_seconds = end-start;
        minTime = std::min(elapsed_seconds.count(), minTime);
    }
    std::cout << "constant memory coefficients and data from shared time " << (blockSize*blocks)/minTime << " elem/s" << " Read BW " << (21*blockSize*blocks*sizeof(float)/1000.0/1000.0/1000.0 )/minTime << " GB/s" <<   std::endl;
    minTime = 10000;
        minTime = 10000;
            for(int i  = 0; i < 100; i++)
    {
        hipDeviceSynchronize(); 
       
        std::chrono::time_point<std::chrono::system_clock> start, end;
        start = std::chrono::system_clock::now();
        stencilShared2<<<blocks, blockSize>>>(a, b, 10240000);
        hipDeviceSynchronize();   
        end = std::chrono::system_clock::now();
       
        getErrorCuda(hipMemcpy(bOut, b, sizeof(float)*10240000, hipMemcpyDefault));
        verify(bOut, bCorr, 1000); 
        std::chrono::duration<float> elapsed_seconds = end-start;
        minTime = std::min(elapsed_seconds.count(), minTime);
    }
    std::cout << "constant memory coefficients and data from shared thorugh read only time " << (blockSize*blocks)/minTime << " elem/s" << " Read BW " << (21*blockSize*blocks*sizeof(float)/1000.0/1000.0/1000.0 )/minTime << " GB/s" <<   std::endl;
    minTime = 10000;


}
